#include "hip/hip_runtime.h"
#include "dataset.h"
#include "seqalgo.h"

using namespace std;

string output_file = "frequent_itemsets.txt";
string output_file_naive = "frequent_itemsets_naive.txt";

__global__ void cuda_compute_li(int *dev_inp_li, int *dev_inp_txids, int *dev_out_li, int *dev_out_txids, int len_eq_class, int total_items, int total_tx, int Th, int level)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x; // tid
	//get the pairs to work on from tid
	//do the interesection of txids and see if the count is > Th
	// if Yes, add the next li and next txids in the output array
	//txid1 is dev_inp_txids[id*NUM_TX]
	int index1 = 0;
	int index2 = 0;
	int cnt = 0;
	int flag = 0;
	for(int i = 0; i < len_eq_class - 1; i++){
		for(int j = i+1; j <= len_eq_class -1; j++){
			if(cnt == id){
				index1 = i;
				index2 = j;
				flag = 1;
				break;
			}
			cnt++;
		}
		if(flag == 1){
			break;
		}
	}

	//printf("For tid: %d, the index1 = %d, index2 = %d \n", id, index1, index2);

	if(index1 == index2){
		//printf("LEAVING: index1 = %d, index2 = %d\n",index1, index2);
		return;
	}

	
	//printf("My candidates are: (%d %d) and (%d %d) \n", dev_inp_li[index1*(level-1)], dev_inp_li[index1*(level-1) + 1], dev_inp_li[index2 * (level - 1)], dev_inp_li[index2 * (level - 1) +1]);
	/*
	printf("txid1: ");
	for(int i = 0; i < NUM_TX; i++){
		if(dev_inp_txids[index1*NUM_TX + i] != -1){
			printf("%d \t", dev_inp_txids[index1*NUM_TX + i]);
		}
		else{
			break;
		}
	}
	printf("\n txid2: ");
	for(int i = 0; i < NUM_TX; i++){
		if(dev_inp_txids[index2*NUM_TX + i] != -1){
			printf("%d \t", dev_inp_txids[index2*NUM_TX + i]);
		}
		else{
			break;
		}
	}
	*/
	//Step: intersection of txids
	int i_txid = index1*total_tx;
	int k_txid = index2*total_tx;
	cnt = 0;
	while( (i_txid < (index1 + 1)*total_tx) && (k_txid < (index2 + 1)*total_tx) )
	{
		if(dev_inp_txids[i_txid] == -1 || dev_inp_txids[k_txid] == -1){
            break;
        }
		else if(dev_inp_txids[i_txid] == dev_inp_txids[k_txid]){
			//store common element
			dev_out_txids[id*total_tx + cnt] = dev_inp_txids[i_txid];
			cnt++;
			i_txid++;
			k_txid++;
		}

		else if (dev_inp_txids[i_txid] > dev_inp_txids[k_txid]){
			k_txid++;
		}
		else{
			i_txid++;
		}
	}

	if(cnt >= Th){
		//printf("Cnt is :%d, Valid candidates: ", cnt);
		//printf("(%d %d) and (%d %d) \n", dev_inp_li[index1*(level-1)], dev_inp_li[index1*(level-1) + 1], dev_inp_li[index2 * (level - 1)], dev_inp_li[index2 * (level - 1) +1]);
		//Step: UNION
		int gap = level - 1;
		int i_li = index1*gap; // 1st element index in li
		int k_li = index2*gap; // 2nd element index in li
		cnt = 0;
		int anchor_pt_li = id*level;
		while((i_li < (index1+1)*gap) && (k_li < (index2+1)*gap)){ 
			if(dev_inp_li[i_li] == dev_inp_li[k_li]){
				//store common element
				//lv_li_arr[cnt] = dev_inp_li[i_li];
				dev_out_li[anchor_pt_li + cnt] = dev_inp_li[i_li];
				//printf("%d \t", dev_inp_li[i_li]);
				cnt++;
				i_li++;
				k_li++;
			}

			else {
				if(dev_inp_li[i_li] < dev_inp_li[k_li]){
					//lv_li_arr[cnt] = dev_inp_li[i_li];
					dev_out_li[anchor_pt_li + cnt] = dev_inp_li[i_li];
					//printf("%d \t", dev_inp_li[i_li]);
					cnt++;
					i_li++;
				}
				else{
					//lv_li_arr[cnt] = dev_inp_li[k_li];
					dev_out_li[anchor_pt_li + cnt] = dev_inp_li[k_li];
					//printf("%d \t", dev_inp_li[k_li]);
					cnt++;
					k_li++;
				}
			}
		}

		if(i_li < (index1+1)*gap){
			while(i_li < (index1+1)*gap){
				dev_out_li[anchor_pt_li + cnt] = dev_inp_li[i_li];
				//printf("%d \t", dev_inp_li[i_li]);
				cnt++;
				i_li++;

			}
		}
		else{
			while(k_li < (index2+1)*gap){
				dev_out_li[anchor_pt_li + cnt] = dev_inp_li[k_li];
				//printf("%d \t", dev_inp_li[k_li]);
				cnt++;
				k_li++;

			}
		}

	}

}

tuple<vector<vector<int>>, vector<set<int>>, long long> compute_li(vector<vector<int>> li, vector<set<int>> litxids, int offset, int level) {
	vector<int> indexes;
	long long time = 0;
	
	auto start = std::chrono::high_resolution_clock::now();

	indexes.push_back(0);
	for (int i = 0; i < li.size() - 1; i++) {
		bool all_same = true;
		for (int j = 0; j <= offset; j++) {
			if (li[i][j] != li[i + 1][j]) {
				all_same = false;
				break;
			}
		}
		if (!all_same) {
			indexes.push_back(i + 1);
		}
	}
	indexes.push_back(li.size());

	vector<vector<int>> li_next;
	vector<set<int>> litxids_next;
	
	/* printf("Index array for level %d is: \t", level);
	for (int i = 0; i < indexes.size() - 1; i++) {
		printf("%d\t", indexes[i]);
	} */

	/* for (int i = 0; i < indexes.size() - 1; i++) {
		printf("Eq class %d has elements: %d\n", i, indexes[i+1] - indexes[i]);
	} */

	// Send this to the GPU: 
	// 1. Partial index array so that each thread can work on it's candidate
	// 2. Send corresponding li and txids. 
	// 3. Output would be a linear array containing the Lis and Txids for next level

	/*printf("This is how the txids look: \n");
	for(int i = 0; i < litxids.size(); i++){
		set<int>::iterator it;
		int cnt = 0;
		for (it = litxids[i].begin(); it != litxids[i].end(); it++) {
			printf("%d \t", *it);
		}
		printf("\n");
	}*/

	for (int i = 0; i < indexes.size() - 1; i++) {
		int len_eq_class = indexes[i+1] - indexes[i];
		int max_candidates = ((len_eq_class * (len_eq_class - 1)) / 2);
		if(max_candidates > 0){
			int byteLen_eq_class = sizeof(int) * len_eq_class;
			int* eq_class_li; // linear array while li is 2D
			int* eq_class_txids; // linear array while litxids is 2D
			hipMallocManaged(&eq_class_li, byteLen_eq_class * (level - 1));
			hipMallocManaged(&eq_class_txids, byteLen_eq_class * (NUM_TX));
			//printf("Level %d, EQ class : %d, Length of eq class is: %d\n", level, i, len_eq_class);
			//preparing the data to send to GPU
			for(int j = 0; j < len_eq_class; j++){
				int anchor_pt_li = j * (level - 1);
				int anchor_pt_txid = j * (NUM_TX);
				for(int x = 0; x < level; x++){
					eq_class_li[anchor_pt_li + x] = li[indexes[i] + j][x];
				}
				
				for(int x = 0; x < NUM_TX; x++){
					eq_class_txids[anchor_pt_txid + x] = -1;
				}

				set<int>::iterator it;
				int cnt = 0;
				for (it = litxids[indexes[i] + j].begin(); it != litxids[indexes[i] + j].end(); it++) {
					eq_class_txids[anchor_pt_txid + cnt] = *it;
					cnt++;
				}
			}
			int * recv_li;
			int * recv_txids;
			int byteLen_li_out = max_candidates * sizeof(int) * level;
			int byteLen_txid_out = max_candidates * sizeof(int) * NUM_TX;
			hipMallocManaged(&recv_li, byteLen_li_out);
			hipMallocManaged(&recv_txids, byteLen_txid_out);

			for(int j = 0; j < max_candidates; j++){
				for(int x = 0; x < level; x++){
					recv_li[j*level + x] = 0;
				}
				for(int x = 0; x < NUM_TX; x++){
					recv_txids[j*NUM_TX + x] = -1;
				}
			}

			//launch the kernel
			cuda_compute_li<<<max_candidates,1>>>(eq_class_li, eq_class_txids, recv_li,recv_txids, len_eq_class, NUM_ITEMS,NUM_TX, THRESHOLD,level);
			hipDeviceSynchronize();

			//put the GPU computed data into li_next and litxids_next
			//printf("max_candidates are: %d, Got this from the GPU: \n", max_candidates);
			/* for(int j = 0; j < max_candidates; j++){
				for(int x = 0; x < level; x++){
						printf("%d",recv_li[j*level + x] );
					}
				printf("\n");
			}

			for(int j = 0; j < max_candidates; j++){
				for(int x = 0; x < NUM_TX; x++){
						printf("%d",recv_txids[j*NUM_TX + x] );
					}
				printf("\n");
			} */

			for(int j = 0; j < max_candidates; j++){
				if(recv_li[j*level] != 0){
					vector<int> items;
					for(int x = 0; x < level; x++){
						//printf("%d",recv_li[j*level + x] );
						 items.push_back(recv_li[j*level + x]);
					}
					li_next.push_back(items);
					set<int> txids;
					//printf("TXIDS are: \n");
					for(int x = 0; x < NUM_TX; x++){
						if(recv_txids[j*NUM_TX + x] != -1){
							//printf("%d\t",recv_txids[j*NUM_TX + x] );
							txids.insert(recv_txids[j*NUM_TX + x]);
						}
						else{
							break;
						}
					}
					litxids_next.push_back(txids);
				}
			}
			hipFree(eq_class_li);
			hipFree(eq_class_txids);
			hipFree(recv_li);
			hipFree(recv_txids);


		}
	}
	//printf("Length of L%d is: %d\n", level, li_next.size());

	/*
	for (int i = 0; i < indexes.size() - 1; i++) {
		for (int j = indexes[i]; j < indexes[i + 1] - 1; j++) {
			for (int k = j + 1; k <= indexes[i + 1] - 1; k++) {
				set<int> txids;
				set_intersection(litxids[j].begin(), litxids[j].end(), litxids[k].begin(), litxids[k].end(), inserter(txids, txids.begin()));
				if (txids.size() >= THRESHOLD) {
					vector<int> items;
					set_union(li[j].begin(), li[j].end(), li[k].begin(), li[k].end(), back_inserter(items));
					li_next.push_back(items);
					litxids_next.push_back(txids);
				}
			}
		}
	}*/
	
	auto elapsed = std::chrono::high_resolution_clock::now() - start;
	time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

#if INFO
		cout << "l" << level << " ->" << endl;
		for (int i = 0; i < li_next.size(); i++) {
			for (int j = 0; j < li_next[i].size(); j++) {
				cout << li_next[i][j];
			}
			cout << "\t";
		}
		cout << endl;
#endif

#if DEBUG
		cout << "l" << level << " txids ->" << endl;
		for (int i = 0; i < litxids_next.size(); i++) {
			for (int j = 0; j < li_next[i].size(); j++) {
				cout << li_next[i][j];
			}
			cout << " = ";
			set<int>::iterator it;
			for (it = litxids_next[i].begin(); it != litxids_next[i].end(); it++) {
				cout << *it << " ";
			}
			cout << endl;
		}
#endif

	tuple<vector<vector<int>>, vector<set<int>>, long long> ret;
	ret = make_tuple(li_next, litxids_next, time);
	return ret;
}

#if NAIVE_METHOD
tuple<vector<vector<int>>, long long> compute_li_naive(vector<vector<int>> li, int level, int offset, entry *db) {
	vector<int> indexes;
	long long time = 0;
	
	auto start = std::chrono::high_resolution_clock::now();

	indexes.push_back(0);
	for (int i = 0; i < li.size() - 1; i++) {
		bool all_same = true;
		for (int j = 0; j <= offset; j++) {
			if (li[i][j] != li[i + 1][j]) {
				all_same = false;
				break;
			}
		}
		if (!all_same) {
			indexes.push_back(i + 1);
		}
	}
	indexes.push_back(li.size());

	/*
	for (int i = 0; i < indexes.size() - 1; i++) {
		printf("Level: %d, Eq class %d has elements: %d\n", level, i, indexes[i+1] - indexes[i]);
	}
	*/

	vector<vector<int>> li_next;
	for (int i = 0; i < indexes.size() - 1; i++) {
		for (int j = indexes[i]; j < indexes[i + 1] - 1; j++) {
			for (int k = j + 1; k <= indexes[i + 1] - 1; k++) {
				vector<int> items;
				set_union(li[j].begin(), li[j].end(), li[k].begin(), li[k].end(), back_inserter(items));
				int count = 0;
				for (int l = 0; l < NUM_TX; l++) {
					bool all_present = true;
					for (int m = 0; m < items.size(); m++) {
						if (db[l].item_present[items[m] - 1] != 1) {
							all_present = false;
							break;
						}
					}
					if (all_present) {
						count++;
					}
				}
				if (count >= THRESHOLD) {
					li_next.push_back(items);
				}
			}
		}
	}
	
	auto elapsed = std::chrono::high_resolution_clock::now() - start;
	time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

#if INFO
		cout << "l" << level << " ->" << endl;
		for (int i = 0; i < li_next.size(); i++) {
			for (int j = 0; j < li_next[i].size(); j++) {
				cout << li_next[i][j];
			}
			cout << "\t";
		}
		cout << endl;
#endif

	tuple<vector<vector<int>>, long long> ret;
	ret = make_tuple(li_next, time);
	return ret;
}
#endif

int main() {
	entry *db = (entry *) malloc(NUM_TX * sizeof(entry));

	generate_dataset(db, NUM_TX, NUM_ITEMS);

#if DEBUG
	print_dataset(db);
#endif

	vector<string> all_freq_itemsets;
	long long total_time = 0;
#if NAIVE_METHOD
	vector<string> all_freq_itemsets_naive;
	long long total_time_naive = 0;
#endif

	vector<int> l1;

	auto start = std::chrono::high_resolution_clock::now();

	/* Computing l1 by checking if occurences of each item across all
	 * transactions exceed the threshold
	 */
	for (int i = 0; i < NUM_ITEMS; i++) {
		int count = 0;
		for (int j = 0; j < NUM_TX; j++) {
			if (db[j].item_present[i] == 1) {
				count++;
			}
		}
		if (count >= THRESHOLD) {
			l1.push_back(i + 1);
		}
	}

	auto elapsed = std::chrono::high_resolution_clock::now() - start;
	total_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
#if NAIVE_METHOD
	total_time_naive += total_time;
#endif

	for (int i = 0; i < l1.size(); i++) {
		all_freq_itemsets.push_back("{" + to_string(l1[i]) + "}");
#if NAIVE_METHOD
		all_freq_itemsets_naive.push_back("{" + to_string(l1[i]) + "}");
#endif
	}

#if INFO
	cout << "l1 ->" << endl;
	for (int i = 0; i < l1.size(); i++) {
		cout << l1[i] << "\t";
	}
	cout << endl;
#endif

	vector<vector<int>> l2;
	vector<set<int>> newdb;
	
	start = std::chrono::high_resolution_clock::now();

	/* Computing l2 by combining pairs within l1
	 * Restructuring the dataset to the format -> itemset : txids
	 */
	for (int i = 0; i < l1.size(); i++) {
		for (int j = i + 1; j < l1.size(); j++) {
			int count = 0;
			set<int> txids;
			for (int k = 0; k < NUM_TX; k++) {
				if (db[k].item_present[l1[i] - 1] == 1 && db[k].item_present[l1[j] - 1] == 1) {
					txids.insert(k);
					count++;
				}
			}
			if (count >= THRESHOLD) {
				vector<int> items;
				items.push_back(l1[i]);
				items.push_back(l1[j]);
				l2.push_back(items);
				newdb.push_back(txids);
			}
		}
	}
	
	elapsed = std::chrono::high_resolution_clock::now() - start;
	total_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

	for (int i = 0; i < l2.size(); i++) {
		stringstream items_stream;
		copy(l2[i].begin(), l2[i].end(), ostream_iterator<int>(items_stream, ", "));
		all_freq_itemsets.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
	}

#if INFO
	cout << "l2 ->" << endl;
	for (int i = 0; i < l2.size(); i++) {
		cout << l2[i][0] << l2[i][1] << "\t";
	}
	cout << endl;
#endif

#if DEBUG
	cout << "Reformed dataset ->" << endl;
	for (int i = 0; i < newdb.size(); i++) {
		cout << l2[i][0] << l2[i][1] << " = ";
		set<int>::iterator it;
		for (it = newdb[i].begin(); it != newdb[i].end(); it++) {
			cout << *it << " ";
		}
		cout << endl;
	}
#endif

	/* Computing li after l2 for all i up to the permitted level
	 */
	vector<vector<int>> li = l2;
	vector<set<int>> litxids = newdb;
	int offset = 0;
	for (int i = 3; i <= LEVEL; i++) {
		if (li.size() >= 1) {
			tuple<vector<vector<int>>, vector<set<int>>, long long> ret = compute_li(li, litxids, offset, i);
			li = get<0>(ret);
			litxids = get<1>(ret);
			total_time += get<2>(ret);
			for (int j = 0; j < li.size(); j++) {
				stringstream items_stream;
				copy(li[j].begin(), li[j].end(), ostream_iterator<int>(items_stream, ", "));
				all_freq_itemsets.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
			}
			offset++;
		}
		else {
			break;
		}
	}

	fstream out_file;
	out_file.open(output_file, fstream::out | fstream::trunc);
	int i;
#if INFO
	cout << "Frequent itemsets ->" << endl;
#endif
	for (i = 0; i < all_freq_itemsets.size() - 1; i++) {
#if INFO
		cout << all_freq_itemsets[i] << "\t";
#endif
		out_file << all_freq_itemsets[i] << endl;
	}
#if INFO
	cout << all_freq_itemsets[i] << endl;
#endif
	out_file << all_freq_itemsets[i] << endl;
	out_file.close();

	cout << "Total execution time (optimal algorithm) = " << total_time << " us" << endl;

#if NAIVE_METHOD
	vector<vector<int>> l2_naive;
	
	start = std::chrono::high_resolution_clock::now();

	/* Computing l2 by combining pairs within l1
	 * No restructuring of the original dataset required in the naive case
	 */
	for (int i = 0; i < l1.size(); i++) {
		for (int j = i + 1; j < l1.size(); j++) {
			int count = 0;
			for (int k = 0; k < NUM_TX; k++) {
				if (db[k].item_present[l1[i] - 1] == 1 && db[k].item_present[l1[j] - 1] == 1) {
					count++;
				}
			}
			if (count >= THRESHOLD) {
				vector<int> items;
				items.push_back(l1[i]);
				items.push_back(l1[j]);
				l2_naive.push_back(items);
			}
		}
	}
	
	elapsed = std::chrono::high_resolution_clock::now() - start;
	total_time_naive += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

	for (int i = 0; i < l2_naive.size(); i++) {
		stringstream items_stream;
		copy(l2_naive[i].begin(), l2_naive[i].end(), ostream_iterator<int>(items_stream, ", "));
		all_freq_itemsets_naive.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
	}

#if INFO
	cout << "l2 ->" << endl;
	for (int i = 0; i < l2_naive.size(); i++) {
		cout << l2_naive[i][0] << l2_naive[i][1] << "\t";
	}
	cout << endl;
#endif

	/* Computing li after l2 for all i up to the permitted level
	 * Always use the original dataset
	 */
	vector<vector<int>> li_naive = l2_naive;
	offset = 0;
	for (int i = 3; i <= LEVEL; i++) {
		if (li_naive.size() >= 1) {
			tuple<vector<vector<int>>, long long> ret = compute_li_naive(li_naive, i, offset, db);
			li_naive = get<0>(ret);
			total_time_naive += get<1>(ret);
			for (int j = 0; j < li_naive.size(); j++) {
				stringstream items_stream;
				copy(li_naive[j].begin(), li_naive[j].end(), ostream_iterator<int>(items_stream, ", "));
				all_freq_itemsets_naive.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
			}
			offset++;
		}
		else {
			break;
		}
	}
	
	fstream out_file_naive;
	out_file_naive.open(output_file_naive, fstream::out | fstream::trunc);
#if INFO
	cout << "Frequent itemsets ->" << endl;
#endif
	for (i = 0; i < all_freq_itemsets_naive.size() - 1; i++) {
#if INFO
		cout << all_freq_itemsets_naive[i] << "\t";
#endif
		out_file_naive << all_freq_itemsets_naive[i] << endl;
	}
#if INFO
	cout << all_freq_itemsets_naive[i] << endl;
#endif
	out_file_naive << all_freq_itemsets_naive[i] << endl;
	out_file_naive.close();

	cout << "Total execution time (naive algorithm) = " << total_time_naive << " us" << endl;
#endif

	return 0;
}
