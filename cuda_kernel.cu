#include "hip/hip_runtime.h"
//convert vector to array - linearizing the vector of vector
int byteLen_li = sizeof(int) * li.size() * (level-1); // total integers stored in the Itemset vector
int * lv_arr = (int *)malloc(byteLen_li);

for(int i = 0; i < li.size(); i++){
    for(int j = 0; j < level - 1; j++){
        int anchor_pt = i*(level-1)
        lv_arr[anchor_pt + j] = li[i][j];
    }
}

//convert set to array - linearizing the vector of set
int byteLen_txid = sizeof(int) * litxids.size() * (NUM_ITEMS); // total integers stored in the Itemset vector
int * lv_arr2 = (int *)malloc(byteLen_txid);

for(int i = 0; i < litxids.size(); i++){
    for(int j = 0; j < NUM_ITEMS; j++){
        int anchor_pt = i*(NUM_ITEMS)
        lv_arr[anchor_pt + j] = litxids[i][j];
    }
}


//create the input and output arrays and do cuda malloc
int *dev_inp_li, *dev_inp_txids, *dev_out_li, *dev_out_txids;
hipMalloc((void**)&dev_inp_li, byteLen_li);
hipMalloc((void**)&dev_inp_txids, byteLen_txid);
int byteLen_li_out = ((li.size() * (li.size() - 1)) / 2) * sizeof(int);
hipMalloc((void**)&dev_out_li, byteLen_out); // len is decided by max combinations which comes from len of li = (len_li) * (len_li-1) * 0.5
int byteLen_txid_out = ((li.size() * (li.size() - 1)) / 2) * sizeof(int) * NUM_ITEMS;
hipMalloc((void**)&dev_out_txids, byteLen_out);
