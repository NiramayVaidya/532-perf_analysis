#include "hip/hip_runtime.h"
#include "dataset.h"
#include "seqalgo.h"
#include <stdio.h>
//#include <hip/hip_runtime_api.h>
//#include <hip/hip_runtime.h>

using namespace std;

string output_file = "frequent_itemsets.txt";
string output_file_naive = "frequent_itemsets_naive.txt";

//int *dev_inp_li, *dev_inp_txids, *dev_inp_indices, *dev_out_li, *dev_out_txids;
// CUDA Kernel
// Each eq class decided by indices array is parallelized. 
__global__ void cuda_compute_li(int *dev_inp_li, int *dev_inp_txids, int *dev_inp_indices, int *dev_out_li, int *dev_out_txids, int index_len, int total_items, int total_tx, int Th, int level)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    printf("-------------------------\n\n");
    printf("INSIDE CUDA: MY TID IS: %d\n", id);
    if(id < index_len - 1){
        //id becomes the index position
        //Step1: find the pairs of next Li
        //Step2: compare with threshold - implement intersection
        //Step3: create the corresponding txids
        //eg: [0,3,5,6] - pairs would be 01, 02, 12
        
        printf("indices array: >>>>>>>>>>>> \n");
        for(int x = 0; x < index_len; x++){
            printf("%d \t", dev_inp_indices[x]);
        }
        printf("End of indices array check\n");
        int start_index = dev_inp_indices[id];
        int end_index = dev_inp_indices[id+1];
        //finding pairs
        for(int i = start_index; i < end_index - 1; i++){
            //within this eq class, find the intersection of txids and count the occurence
            for(int k = i + 1; k <= end_index - 1; k++){
                //check for i,k pair
                int i_txid = i*total_tx;
                int k_txid = k*total_tx;
                int cnt = 0;
                int lv_arr[NUM_TX];
                for(int i = 0; i < total_tx; i++){
                    lv_arr[i] = -1;
                }

                //TODO print the corresponding data: 
                //printf(                      );
                for(int x = 0; x < level-1; x++){
                    printf("");
                }
                
                //int lv_it_k = k_txid;
                /*
                printf("First element's TXIDS\n");
                for(int x = 0; x < total_tx; x++){
                    printf("%d\t", dev_inp_txids[i_txid + x]);
                }

                
                printf("\n Second element's TXIDS\n");
                for(int x = 0; x < total_tx; x++){
                    printf("%d\t", dev_inp_txids[k_txid + x]);
                }
                printf("\n");
                */
                //intersection
                while((i_txid < (i+1)*total_tx) && (k_txid < (k+1)*total_tx)){
                    if(dev_inp_txids[i_txid] == -1 || dev_inp_txids[k_txid] == -1){
                        break;
                    }

                    else if(dev_inp_txids[i_txid] == dev_inp_txids[k_txid]){
                        //store common element
                        lv_arr[cnt] = dev_inp_txids[i_txid];
                        cnt++;
                        i_txid++;
                        k_txid++;
                    }

                    else if (dev_inp_txids[i_txid] > dev_inp_txids[k_txid]){
                        k_txid++;
                    }
                    else{
                        i_txid++;
                    }

                }
                
                //check if count > Th
                if(cnt >= Th){
                    //do union and copy the intersection wala txid
                    int gap = level - 1;
                    int i_li = i*gap; // 1st element index in li
                    int k_li = k*gap; // 2nd element index in li
                    //int lv_li_arr[level];
                    printf("Value of cnt is: %d for pair: (%d %d), (%d %d)\n", cnt,dev_inp_li[i_li], dev_inp_li[i_li + 1], dev_inp_li[k_li], dev_inp_li[k_li+ 1] );

                    int anchor_pt_li = (i+k)*level;
                    cnt = 0;
                    printf("INSIDE CUDA: printing the next li: \n");
                    //union
                    while((i_li < (i+1)*gap) && (k_li < (k+1)*gap)){
                        
                        if(dev_inp_li[i_li] == dev_inp_li[k_li]){
                            //store common element
                            //lv_li_arr[cnt] = dev_inp_li[i_li];
                            dev_out_li[anchor_pt_li + cnt] = dev_inp_li[i_li];
                            printf("%d \t", dev_inp_li[i_li]);
                            cnt++;
                            i_li++;
                            k_li++;
                        }

                        else {
                            if(dev_inp_li[i_li] < dev_inp_li[k_li]){
                                //lv_li_arr[cnt] = dev_inp_li[i_li];
                                dev_out_li[anchor_pt_li + cnt] = dev_inp_li[i_li];
                                printf("%d \t", dev_inp_li[i_li]);
                                cnt++;
                                i_li++;
                            }
                            else{
                                //lv_li_arr[cnt] = dev_inp_li[k_li];
                                dev_out_li[anchor_pt_li + cnt] = dev_inp_li[k_li];
                                printf("%d \t", dev_inp_li[k_li]);
                                cnt++;
                                k_li++;
                            }
                        }
                    }
                    if(i_li < (i+1)*gap){
                        while(i_li < (i+1)*gap){
                            dev_out_li[anchor_pt_li + cnt] = dev_inp_li[i_li];
                            printf("%d \t", dev_inp_li[i_li]);
                            cnt++;
                            i_li++;

                        }
                    }
                    else{
                        while(k_li < (k+1)*gap){
                            dev_out_li[anchor_pt_li + cnt] = dev_inp_li[k_li];
                            printf("%d \t", dev_inp_li[k_li]);
                            cnt++;
                            k_li++;

                        }
                    }
                    printf("\n");
                    
                    // store the txids and union in the output array
                    /*
                    for(int z = 0; z < level; z++){
                        dev_out_li[anchor_pt_li + z] = lv_li_arr[z];
                    }*/

                    int anchor_pt_txid = (i+k)*total_tx;
                    printf("INSIDE CUDA: printing the next txids: \n");
                    for(int z = 0; z < total_tx; z++){
                        dev_out_txids[anchor_pt_txid + z] = lv_arr[z];
                        printf("%d \t", lv_arr[z]);
                    }
                    printf("\n");

            }
        }
    }
}

    printf("-------------------------\n\n");
}

tuple<vector<vector<int>>, vector<set<int>>, long long> compute_li(vector<vector<int>> li, vector<set<int>> litxids, int offset, int level) {
	vector<int> indexes;
	long long time = 0;
	
	auto start = std::chrono::high_resolution_clock::now();

	indexes.push_back(0);
	for (int i = 0; i < li.size() - 1; i++) {
		bool all_same = true;
		for (int j = 0; j <= offset; j++) {
			if (li[i][j] != li[i + 1][j]) {
				all_same = false;
				break;
			}
		}
		if (!all_same) {
			indexes.push_back(i + 1);
		}
	}

    //print indices
    
    indexes.push_back(li.size());
    vector<int>::iterator ptr;
    printf("Indices for level: %d\n ", level);
    for (ptr = indexes.begin(); ptr < indexes.end(); ptr++){
        cout << *ptr << " ";
    }
	printf("\n");
    

    //Inputs here are: 
    //indexes array: [0,2,4]
    //L2: [{1, 5}	{1, 6}	{3, 5}	{3, 6}	{5, 6}]
    //txids:[[], [], [], [], []] - the number of elements in L2 and txids should be same, like index of AB is L2 should also point to its corresponding txids. 
    //To do work with GPU: 
    //INPUT to GPU: index array, L2, txids
    //OUTPUT from GPU: L3, txids
    //convert L2 vector to linear array - each itemset element separated by the level (3 means 2 len itemset)
    //convert txid set to linear array- each entry separated by len of dataset.
    //have the corresponding output arrays. 
	
    //convert vector to array - linearizing the vector of vector
    int byteLen_li = sizeof(int) * li.size() * (level-1); // total integers stored in the Itemset vector
    int * lv_arr = (int *)malloc(byteLen_li);

    for(int i = 0; i < li.size(); i++){
        for(int j = 0; j < level - 1; j++){
            int anchor_pt = i*(level-1);
            lv_arr[anchor_pt + j] = li[i][j];
            //printf("%d \t ", li[i][j]);
        }
       // printf("\n");
    }

    int byteLen_indices = sizeof(int) * indexes.size();
    int * lv_index_arr = (int*)malloc(byteLen_indices);
    for(int i = 0; i < indexes.size(); i++){
            lv_index_arr[i] = indexes[i];
    }



    //convert set to array - linearizing the vector of set
    int byteLen_txid = sizeof(int) * litxids.size() * (NUM_TX); // total integers stored in set (max)
    int * lv_arr2 = (int *)malloc(byteLen_txid);

    for(int i = 0; i < litxids.size(); i++){
        int anchor_pt = i*(NUM_TX);
        for(int j = 0; j < NUM_TX; j++){
            lv_arr2[anchor_pt + j] = -1;
        }
        
        set<int>::iterator it;
        int cnt = 0;
		for (it = litxids[i].begin(); it != litxids[i].end(); it++) {
            lv_arr2[anchor_pt + cnt] = *it;
            cnt++;
		}

    }
   /* 
    printf("The corresponding tTXIDS are: \n");

    for(int i = 0; i < litxids.size(); i++){
        for(int j = 0; j < NUM_TX; j++){
            int anchor_pt = i*(NUM_TX);
            printf("%d \t", lv_arr2[anchor_pt + j]);
        }
        printf("\n");
        
    }

    printf("Total elements in Li are: %d\n", li.size());
    printf("Total elements in Txid are: %d\n", litxids.size());
    
*/

    //create the input and output arrays and do cuda malloc
    int *dev_inp_li, *dev_inp_txids, *dev_inp_indices, *dev_out_li, *dev_out_txids;
    hipMalloc((void**)&dev_inp_li, byteLen_li);
    hipMalloc((void**)&dev_inp_txids, byteLen_txid);
    hipMalloc((void**)&dev_inp_indices, byteLen_indices);
    int byteLen_li_out = ((li.size() * (li.size() - 1)) / 2) * sizeof(int) * level;
    hipMalloc((void**)&dev_out_li, byteLen_li_out); // len is decided by max combinations which comes from len of li = (len_li) * (len_li-1) * 0.5
    int byteLen_txid_out = ((li.size() * (li.size() - 1)) / 2) * sizeof(int) * NUM_TX;
    hipMalloc((void**)&dev_out_txids, byteLen_txid_out);
    
    int * recv_li = (int *)malloc(byteLen_li_out);
    int * recv_txids = (int *)malloc(byteLen_txid_out);
    hipMemcpy(dev_inp_li,lv_arr , byteLen_li, hipMemcpyHostToDevice);
    hipMemcpy(dev_inp_txids, lv_arr2, byteLen_txid, hipMemcpyHostToDevice);
    hipMemcpy(dev_inp_indices, lv_index_arr, byteLen_indices, hipMemcpyHostToDevice);

    int len_indices = indexes.size();
    printf("Launching kernel!\n");
    //TODO change total TBs to #eq_classes
    cuda_compute_li<<<1,1>>>(dev_inp_li, dev_inp_txids, dev_inp_indices, dev_out_li,dev_out_txids, len_indices, NUM_ITEMS,NUM_TX, THRESHOLD,level);
    hipMemcpy(recv_li, dev_out_li, byteLen_li_out, hipMemcpyDeviceToHost);
    hipMemcpy(recv_txids, dev_out_txids,byteLen_txid_out, hipMemcpyDeviceToHost);
    


    //print recevied li
    printf("##### GPU computed Lis are: #####\n");
    for(int i = 0; i < byteLen_li_out/sizeof(int); i = i + level){
        for(int j = 0; j < level; j++){
            int anchor_pt = i*(level);
            //lv_arr[anchor_pt + j] = li[i][j];
            printf("%d \t ", recv_li[anchor_pt + j]);
        }
        printf("\n");
    }
    
    printf("##### GPU computed TXIDs are: #####\n");
    for(int i = 0; i < byteLen_txid_out/sizeof(int); i = i + NUM_TX){
        for(int j = 0; j < NUM_TX; j++){
            int anchor_pt = i*(NUM_TX);
            //lv_arr[anchor_pt + j] = li[i][j];
            printf("%d \t ", recv_txids[anchor_pt + j]);
        }
        printf("\n");
    }

    vector<vector<int>> li_next;
	vector<set<int>> litxids_next;
	/*
    for (int i = 0; i < indexes.size() - 1; i++) {
		for (int j = indexes[i]; j < indexes[i + 1] - 1; j++) {
			for (int k = j + 1; k <= indexes[i + 1] - 1; k++) {
				set<int> txids;
				set_intersection(litxids[j].begin(), litxids[j].end(), litxids[k].begin(), litxids[k].end(), inserter(txids, txids.begin()));
				if (txids.size() >= THRESHOLD) {
					vector<int> items;
					set_union(li[j].begin(), li[j].end(), li[k].begin(), li[k].end(), back_inserter(items));
					li_next.push_back(items);
					litxids_next.push_back(txids);
				}
			}
		}
	}*/
	
	auto elapsed = std::chrono::high_resolution_clock::now() - start;
	time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

#if INFO
		cout << "l" << level << " ->" << endl;
		for (int i = 0; i < li_next.size(); i++) {
			for (int j = 0; j < li_next[i].size(); j++) {
				cout << li_next[i][j];
			}
			cout << "\t";
		}
		cout << endl;
#endif

#if DEBUG
		cout << "l" << level << " txids ->" << endl;
		for (int i = 0; i < litxids_next.size(); i++) {
			for (int j = 0; j < li_next[i].size(); j++) {
				cout << li_next[i][j];
			}
			cout << " = ";
			set<int>::iterator it;
			for (it = litxids_next[i].begin(); it != litxids_next[i].end(); it++) {
				cout << *it << " ";
			}
			cout << endl;
		}
#endif

	tuple<vector<vector<int>>, vector<set<int>>, long long> ret;
	ret = make_tuple(li_next, litxids_next, time);
	return ret;
}

#if NAIVE_METHOD
tuple<vector<vector<int>>, long long> compute_li_naive(vector<vector<int>> li, int level, int offset, entry *db) {
	vector<int> indexes;
	long long time = 0;
	
	auto start = std::chrono::high_resolution_clock::now();

	indexes.push_back(0);
	for (int i = 0; i < li.size() - 1; i++) {
		bool all_same = true;
		for (int j = 0; j <= offset; j++) {
			if (li[i][j] != li[i + 1][j]) {
				all_same = false;
				break;
			}
		}
		if (!all_same) {
			indexes.push_back(i + 1);
		}
	}
	indexes.push_back(li.size());

	vector<vector<int>> li_next;
	for (int i = 0; i < indexes.size() - 1; i++) {
		for (int j = indexes[i]; j < indexes[i + 1] - 1; j++) {
			for (int k = j + 1; k <= indexes[i + 1] - 1; k++) {
				vector<int> items;
				set_union(li[j].begin(), li[j].end(), li[k].begin(), li[k].end(), back_inserter(items));
				int count = 0;
				for (int l = 0; l < NUM_TX; l++) {
					bool all_present = true;
					for (int m = 0; m < items.size(); m++) {
						if (db[l].item_present[items[m] - 1] != 1) {
							all_present = false;
							break;
						}
					}
					if (all_present) {
						count++;
					}
				}
				if (count >= THRESHOLD) {
					li_next.push_back(items);
				}
			}
		}
	}
	
	auto elapsed = std::chrono::high_resolution_clock::now() - start;
	time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

#if INFO
		cout << "l" << level << " ->" << endl;
		for (int i = 0; i < li_next.size(); i++) {
			for (int j = 0; j < li_next[i].size(); j++) {
				cout << li_next[i][j];
			}
			cout << "\t";
		}
		cout << endl;
#endif

	tuple<vector<vector<int>>, long long> ret;
	ret = make_tuple(li_next, time);
	return ret;
}
#endif

int main() {
	entry *db = (entry *) malloc(NUM_TX * sizeof(entry));

	generate_dataset(db, NUM_TX, NUM_ITEMS);

	print_dataset(db);
#if DEBUG
#endif

	vector<string> all_freq_itemsets;
	long long total_time = 0;
#if NAIVE_METHOD
	vector<string> all_freq_itemsets_naive;
	long long total_time_naive = 0;
#endif

	vector<int> l1;

	auto start = std::chrono::high_resolution_clock::now();

	/* Computing l1 by checking if occurences of each item across all
	 * transactions exceed the threshold
	 */
	for (int i = 0; i < NUM_ITEMS; i++) {
		int count = 0;
		for (int j = 0; j < NUM_TX; j++) {
			if (db[j].item_present[i] == 1) {
				count++;
			}
		}
		if (count >= THRESHOLD) {
			l1.push_back(i + 1);
		}
	}

	auto elapsed = std::chrono::high_resolution_clock::now() - start;
	total_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();
#if NAIVE_METHOD
	total_time_naive += total_time;
#endif

	for (int i = 0; i < l1.size(); i++) {
		all_freq_itemsets.push_back("{" + to_string(l1[i]) + "}");
#if NAIVE_METHOD
		all_freq_itemsets_naive.push_back("{" + to_string(l1[i]) + "}");
#endif
	}

#if INFO
	cout << "l1 ->" << endl;
	for (int i = 0; i < l1.size(); i++) {
		cout << l1[i] << "\t";
	}
	cout << endl;
#endif

	vector<vector<int>> l2;
	vector<set<int>> newdb;
	
	start = std::chrono::high_resolution_clock::now();

	/* Computing l2 by combining pairs within l1
	 * Restructuring the dataset to the format -> itemset : txids
	 */
	for (int i = 0; i < l1.size(); i++) {
		for (int j = i + 1; j < l1.size(); j++) {
			int count = 0;
			set<int> txids;
			for (int k = 0; k < NUM_TX; k++) {
				if (db[k].item_present[l1[i] - 1] == 1 && db[k].item_present[l1[j] - 1] == 1) {
					txids.insert(k);
					count++;
				}
			}
			if (count >= THRESHOLD) {
				vector<int> items;
				items.push_back(l1[i]);
				items.push_back(l1[j]);
				l2.push_back(items);
				newdb.push_back(txids);
			}
		}
	}
	
	elapsed = std::chrono::high_resolution_clock::now() - start;
	total_time += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

	for (int i = 0; i < l2.size(); i++) {
		stringstream items_stream;
		copy(l2[i].begin(), l2[i].end(), ostream_iterator<int>(items_stream, ", "));
		all_freq_itemsets.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
	}

#if INFO
	cout << "l2 ->" << endl;
	for (int i = 0; i < l2.size(); i++) {
		cout << l2[i][0] << l2[i][1] << "\t";
	}
	cout << endl;
#endif

#if DEBUG
	cout << "Reformed dataset ->" << endl;
	for (int i = 0; i < newdb.size(); i++) {
		cout << l2[i][0] << l2[i][1] << " = ";
		set<int>::iterator it;
		for (it = newdb[i].begin(); it != newdb[i].end(); it++) {
			cout << *it << " ";
		}
		cout << endl;
	}
#endif

	/* Computing li after l2 for all i up to the permitted level
	 */
	vector<vector<int>> li = l2;
	vector<set<int>> litxids = newdb;
	int offset = 0;
	for (int i = 3; i <= LEVEL; i++) {
		if (li.size() >= 1) {
			tuple<vector<vector<int>>, vector<set<int>>, long long> ret = compute_li(li, litxids, offset, i);
			li = get<0>(ret);
			litxids = get<1>(ret);
			total_time += get<2>(ret);
			for (int j = 0; j < li.size(); j++) {
				stringstream items_stream;
				copy(li[j].begin(), li[j].end(), ostream_iterator<int>(items_stream, ", "));
				all_freq_itemsets.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
			}
			offset++;
		}
		else {
			break;
		}
	}

	fstream out_file;
	out_file.open(output_file, fstream::out | fstream::trunc);
	int i;
#if INFO
	cout << "Frequent itemsets ->" << endl;
#endif
	for (i = 0; i < all_freq_itemsets.size() - 1; i++) {
#if INFO
		cout << all_freq_itemsets[i] << "\t";
#endif
		out_file << all_freq_itemsets[i] << endl;
	}
#if INFO
	cout << all_freq_itemsets[i] << endl;
#endif
	out_file << all_freq_itemsets[i] << endl;
	out_file.close();

	cout << "Total execution time (optimal algorithm) = " << total_time << " us" << endl;

#if NAIVE_METHOD
	vector<vector<int>> l2_naive;
	
	start = std::chrono::high_resolution_clock::now();

	/* Computing l2 by combining pairs within l1
	 * No restructuring of the original dataset required in the naive case
	 */
	for (int i = 0; i < l1.size(); i++) {
		for (int j = i + 1; j < l1.size(); j++) {
			int count = 0;
			for (int k = 0; k < NUM_TX; k++) {
				if (db[k].item_present[l1[i] - 1] == 1 && db[k].item_present[l1[j] - 1] == 1) {
					count++;
				}
			}
			if (count >= THRESHOLD) {
				vector<int> items;
				items.push_back(l1[i]);
				items.push_back(l1[j]);
				l2_naive.push_back(items);
			}
		}
	}
	
	elapsed = std::chrono::high_resolution_clock::now() - start;
	total_time_naive += std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

	for (int i = 0; i < l2_naive.size(); i++) {
		stringstream items_stream;
		copy(l2_naive[i].begin(), l2_naive[i].end(), ostream_iterator<int>(items_stream, ", "));
		all_freq_itemsets_naive.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
	}

#if INFO
	cout << "l2 ->" << endl;
	for (int i = 0; i < l2_naive.size(); i++) {
		cout << l2_naive[i][0] << l2_naive[i][1] << "\t";
	}
	cout << endl;
#endif

	/* Computing li after l2 for all i up to the permitted level
	 * Always use the original dataset
	 */
	vector<vector<int>> li_naive = l2_naive;
	offset = 0;
	for (int i = 3; i <= LEVEL; i++) {
		if (li_naive.size() >= 1) {
			tuple<vector<vector<int>>, long long> ret = compute_li_naive(li_naive, i, offset, db);
			li_naive = get<0>(ret);
			total_time_naive += get<1>(ret);
			for (int j = 0; j < li_naive.size(); j++) {
				stringstream items_stream;
				copy(li_naive[j].begin(), li_naive[j].end(), ostream_iterator<int>(items_stream, ", "));
				all_freq_itemsets_naive.push_back("{" + items_stream.str().substr(0, items_stream.str().size() - 2) + "}");
			}
			offset++;
		}
		else {
			break;
		}
	}
	
	fstream out_file_naive;
	out_file_naive.open(output_file_naive, fstream::out | fstream::trunc);
#if INFO
	cout << "Frequent itemsets ->" << endl;
#endif
	for (i = 0; i < all_freq_itemsets_naive.size() - 1; i++) {
#if INFO
		cout << all_freq_itemsets_naive[i] << "\t";
#endif
		out_file_naive << all_freq_itemsets_naive[i] << endl;
	}
#if INFO
	cout << all_freq_itemsets_naive[i] << endl;
#endif
	out_file_naive << all_freq_itemsets_naive[i] << endl;
	out_file_naive.close();

	cout << "Total execution time (naive algorithm) = " << total_time_naive << " us" << endl;
#endif

	return 0;
}
